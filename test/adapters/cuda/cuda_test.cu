#include "hip/hip_runtime.h"
/*
 * This file is part of the Score-P software (http://www.score-p.org)
 *
 * Copyright (c) 2009-2013,
 *    RWTH Aachen University, Germany
 *    Gesellschaft fuer numerische Simulation mbH Braunschweig, Germany
 *    Technische Universitaet Dresden, Germany
 *    University of Oregon, Eugene, USA
 *    Forschungszentrum Juelich GmbH, Germany
 *    German Research School for Simulation Sciences GmbH, Juelich/Aachen, Germany
 *    Technische Universitaet Muenchen, Germany
 *
 * See the COPYING file in the package base directory for details.
 * 
 */


/**
 * @file       cuda_test.cu
 * @maintainer Robert Dietrich <robert.dietrich@zih.tu-dresden.de>
 *
 * @brief Test program for the CUDA adapter. Several parts of this program have 
 * been extracted from the NVIDIA computing samples 'simpleStreams' and 
 * 'concurrentKernels'
 */

#include <config.h>

#include <stdio.h>
#include <unistd.h>
#include <stdint.h>

// CUDA utilities and system includes
#include <hip/hip_runtime.h>

#define CUDART_CALL(_err, _msg) \
  if(hipSuccess != _err)         \
    __checkCUDACall(_err, _msg, __FILE__,__LINE__)

static uint32_t kernel_workload = 20;

static uint64_t cpu_usleeptime = 20000;

/* function declarations */
static void __checkCUDACall(hipError_t ecode, const char* msg,
                            const char *file, const int line);

static void runCopyComputeOverlap(int nstreams);

static void runConcurrentKernels(int nstreams);

static void show_help(void);

static char getopt(char *argument);

static void setArguments(int argc, char* argv[]);


__global__ void init_array(int *g_data, int *factor, int num_iterations)
{ 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  for(int i=0;i<num_iterations;i++)
    g_data[idx] += *factor; // non-coalesced on purpose, to burn time
}

// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clock_block(clock_t* d_o, clock_t clock_count)
{ 
	clock_t start_clock = clock();
	
	clock_t clock_offset = 0;

	while( clock_offset < clock_count ) {
		clock_offset = clock() - start_clock;
	}

	d_o[0] = clock_offset;
}

int main(int argc, char **argv)
{
  int nstreams = 3; // number of streams for CUDA calls
  
  // check the compute capability of the device
  int num_devices=0;

  setArguments(argc, argv);

  CUDART_CALL(hipGetDeviceCount(&num_devices), "hipGetDeviceCount");
  if(0==num_devices){
    printf("your system does not have a CUDA capable device, waiving test...\n");
    exit(-1);
  }

  /* check if the command-line chosen device ID is within range, exit if not
  if( cuda_device >= num_devices ){
    printf("cuda_device=%d is invalid, must choose device ID between 0 and %d\n", cuda_device, num_devices-1);
    exit(-1);
  }*/

  runCopyComputeOverlap(nstreams);
  
  runConcurrentKernels(nstreams);
}

static void runCopyComputeOverlap(int nstreams)
{
	int nreps = 3;                 // number of times each experiment is repeated
	int n = 16 * 1024 * 1024;       // number of integers in the data set
	int nbytes = n * sizeof(int);   // number of data bytes
	dim3 threads, blocks;           // kernel launch configuration
  int niterations = kernel_workload;	// number of iterations for the loop inside the kernel_time
  
	// allocate host memory
	int c = 5;            // value to which the array will be initialized
	int *h_a = 0;         // pointer to the array data in host memory
	int *hAligned_a = 0;  // pointer to the array data in host memory (aligned to MEMORY_ALIGNMENT)

	// allocate host memory (pinned is required for achieve asynchronicity)
	CUDART_CALL(hipHostMalloc((void**)&h_a, nbytes), "hipHostMalloc"); 
	hAligned_a = h_a; 

	// allocate device memory
	int *d_a = 0, *d_c = 0;             // pointers to data and init value in the device memory
	CUDART_CALL( hipMalloc((void**)&d_a, nbytes), "hipMalloc");
	CUDART_CALL( hipMalloc((void**)&d_c, sizeof(int)), "hipMalloc");
	CUDART_CALL( hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice), "hipMemcpy");
	
	threads=dim3(512, 1);
  blocks=dim3(n / threads.x, 1);
	init_array<<<blocks, threads>>>(d_a, d_c, niterations);
	usleep(cpu_usleeptime);
	hipMemcpyAsync(hAligned_a, d_a, nbytes, hipMemcpyDeviceToHost);

	// allocate and initialize an array of stream handles
	hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
	for(int i = 0; i < nstreams; i++) {
		CUDART_CALL( hipStreamCreate(&(streams[i])), "hipStreamCreate");
	}

  niterations = kernel_workload;
	
	printf("\nStarting Copy/Compute overlap test\n");
	threads=dim3(512,1);
	blocks=dim3(n/(nstreams*threads.x),1);
	memset(hAligned_a, 255, nbytes); // set host memory bits to all 1s, for testing correctness
	hipMemset(d_a, 0, nbytes); // set device memory to all 0s, for testing correctness
	for(int k = 0; k < nreps; k++){
		// asynchronously launch nstreams kernels, each operating on its own portion of data
		for(int i = 0; i < nstreams; i++)
		  init_array<<<blocks, threads, 0, streams[i]>>>(d_a + i * n / nstreams, d_c, niterations);

		// asynchronously launch nstreams memcopies.  Note that memcopy in stream x will only
		//   commence executing when all previous CUDA calls in stream x have completed
		for(int i = 0; i < nstreams; i++)
			hipMemcpyAsync(hAligned_a + i * n / nstreams, d_a + i * n / nstreams, nbytes / nstreams, hipMemcpyDeviceToHost, streams[i]);
	}
	CUDART_CALL(hipDeviceSynchronize(), "hipDeviceSynchronize");
	
	// release resources
	for(int i = 0; i < nstreams; i++) {
		hipStreamDestroy(streams[i]); 
	}
	free(streams);
	
	hipHostFree(h_a);
	hipFree(d_a);
	hipFree(d_c);
}

static void runConcurrentKernels(int nstreams)
{
	float kernel_time = 10; // time the kernel should run in ms
	hipDeviceProp_t deviceProp;
	clock_t *a = NULL;               // pointer to the array data in host memory
	int nbytes = nstreams * sizeof(clock_t);   // number of data bytes
	int cuda_device = 0;
	
	CUDART_CALL(hipGetDevice(&cuda_device), "hipGetDevice");
	
	CUDART_CALL(hipGetDeviceProperties(&deviceProp, cuda_device), "hipGetDeviceProperties");
	if( (deviceProp.concurrentKernels == 0 )) {
		printf("> GPU does not support concurrent kernel execution\n");
		printf("  CUDA kernel runs will be serialized\n");
	}
	
	// allocate host memory
	CUDART_CALL(hipHostMalloc((void**)&a, nbytes), "hipHostMalloc"); 

	// allocate device memory
	clock_t *d_ac = 0;             // pointers to data and init value in the device memory
	CUDART_CALL(hipMalloc((void**)&d_ac, nbytes), "hipMalloc");
	
	// allocate and initialize an array of stream handles
	hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
	for(int i = 0; i < nstreams; i++) {
		CUDART_CALL( hipStreamCreate(&(streams[i])), "hipStreamCreate");
	}
	
	// time execution with nkernels streams
  clock_t total_clocks = 0;
  clock_t time_clocks = kernel_time * deviceProp.clockRate;
	printf("\nStarting concurrent kernel test\n");
	
  // queue nkernels in separate streams and record when they are done
	for( int i=0; i<nstreams; ++i) {
		clock_block<<<1,1,0,streams[i]>>>(&d_ac[i], time_clocks);
		total_clocks += time_clocks;
	}
	
	CUDART_CALL(hipDeviceSynchronize(), "hipDeviceSynchronize");
	
	// release resources
	for(int i = 0; i < nstreams; i++) {
		hipStreamDestroy(streams[i]); 
	}
	free(streams);
	hipHostFree(a);
	hipFree(d_ac);
}

/* 
 * Checks if a CUDA runtime API call returns successful and respectively prints
 * the error.
 * 
 * @param ecode the CUDA error code
 * @param msg a message to get more detailed information about the error
 * @param the corresponding file
 * @param the line the error occurred
 */
static void __checkCUDACall(hipError_t ecode, const char* msg,
                            const char *file, const int line)
{
  if(msg != NULL) printf("[CUDART] %s",  msg);
  printf("[CUDA Error <%s>:%i] %s", file, line, hipGetErrorString(ecode));
}

static void show_help(void) 
{
   printf("\noverhead [OPTION]\n"
          "\t-g  kernel workload as number of loop iterations (positive integer)\n"
          "\t-c  sleep time of host after first kernel launch in seconds (positive integer)\n\n"
         );
}

static char getopt(char *argument) 
{
  if( argument[0]=='-') return argument[1];
  return 'f';
}

static void setArguments(int argc, char* argv[])
{
  int j=1;

  while(j<argc){
    switch(getopt(argv[j])){
      case 'g': // number of loop iterations inside GPU kernel
        kernel_workload = atoi(argv[++j]);
        break;

      case 'c': // seconds to sleep after launch of first kernel
        cpu_usleeptime = atoi(argv[++j])*1000;
        break;

      default:
        show_help();
        exit(1);
    }
    j++;
  }
}
